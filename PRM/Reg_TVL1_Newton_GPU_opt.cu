#include "hip/hip_runtime.h"
/***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <mex.h>
#include <math.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "Reg_TVL1_Newton_kernels.cu"

#define BLOCKSIZE 512
#define MAX_GRID_SIZE 65535
#define NUMTHREADS 512

dim3 GetGrid(int size){
    size = (size-1) / NUMTHREADS + 1;
    dim3 grid( size, 1, 1 );
    if( grid.x > MAX_GRID_SIZE ) grid.x = grid.y = (int) sqrt( (double)(size-1) ) + 1;
    else if( grid.y > MAX_GRID_SIZE ) grid.x = grid.y = grid.z = (int) pow( (double)(size-1), (double)1.0/3.0 ) + 1;
    return grid;
}

extern void mexFunction(int iNbOut, mxArray *pmxOut[],
        int iNbIn, const mxArray *pmxIn[]){
    
    /* iNbOut: number of outputs */
    /* pmxOut: array of pointers to output arguments */
    
    /* iNbIn: number of inputs
    /* pmxIn: array of pointers to input arguments */
    
    /*  host arrays and variables */
    float   *h_ux, *h_uy, *h_uz, *h_cvg, *h_Ux, *h_Uy, *h_Uz;
    float   *h_VecParameters,*h_Gx, *h_Gy, *h_Gz, *h_Gf, *h_Gt;
    float   *h_bx1, *h_bx2, *h_bx3, *h_by1, *h_by2, *h_by3, *h_bz1, *h_bz2, *h_bz3;
    float   *h_q, *h_gkx, *h_gky, *h_gkz, *tt, *h_dvx, *h_dvy, *h_dvz;
    float   fError, cc, steps, fPenalty, fps;
    /*
    int     *punum, iNy, iNx, iNz, iNdim, iDim[3], iNI;
    int     maxIter, SZF, iDev;
    */
    int     *punum, iNy, iNx, iNz, iNdim, iDim[3], maxIter;
    
    
    hipSetDevice(1);
    
    /* Timing */
    hipEvent_t start, stop;
    float time;
    
    /*  device arrays */
    float   *d_bx1, *d_by1, *d_bz1, *d_bx2, *d_by2, *d_bz2, *d_bx3, *d_by3, *d_bz3, *d_dvx;
    float   *d_q, *d_dvy, *d_dvz, *d_gkx, *d_gky, *d_gkz, *d_ux, *d_uy, *d_uz;
    float   *d_Ux, *d_Uy, *d_Uz, *d_Gx, *d_Gy, *d_Gz, *d_Gf, *d_Gt, *h_FPS, *d_FPS;
    
    
    /* CUDA event-based timer start */
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );
    
    /* input interface with matlab arrays */
    h_VecParameters = (float *)mxGetData(pmxIn[0]); /* Vector of parameters */
    h_Ux = (float *)mxGetData(pmxIn[1]);
    h_Uy = (float *)mxGetData(pmxIn[2]);
    h_Uz = (float *)mxGetData(pmxIn[3]);
    h_Gx = (float *)mxGetData(pmxIn[4]);
    h_Gy = (float *)mxGetData(pmxIn[5]);
    h_Gz = (float *)mxGetData(pmxIn[6]);
    h_Gt = (float *)mxGetData(pmxIn[7]);
    h_Gf = (float *)mxGetData(pmxIn[8]);
    
    
    /* dimensions */
    iNy = (int) h_VecParameters[0];
    iNx = (int) h_VecParameters[1];
    iNz = (int) h_VecParameters[2];
    
    unsigned int imageSize = iNx*iNy*iNz;
    
    /* parameters */
    maxIter = (int) h_VecParameters[3]; /* total number of iterations */
    fError = (float) h_VecParameters[4]; /* error criterion */
    cc = (float) h_VecParameters[5]; /* cc for ALM */
    steps = (float) h_VecParameters[6]; /* steps for each iteration */
    fPenalty = (float) h_VecParameters[7];
    
    /* output interface with matlab */
    /* ux */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    pmxOut[0] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_ux = (float*)mxGetData(pmxOut[0]);
    
    /* uy */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    
    pmxOut[1] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_uy = (float*)mxGetData(pmxOut[1]);
    
    /* uz */
    iNdim = 3;
    iDim[0] = iNy;
    iDim[1] = iNx;
    iDim[2] = iNz;
    
    pmxOut[2] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_uz = (float*)mxGetData(pmxOut[2]);
    
    /* convergence rate */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = maxIter;
    pmxOut[3] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    h_cvg = (float*)mxGetData(pmxOut[3]);
    
    /* number of iterations */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = 1;
    pmxOut[4] = mxCreateNumericArray(iNdim,(const int*)iDim,mxUINT16_CLASS,mxREAL);
    punum = (int*)mxGetData(pmxOut[4]);
    
    /* computation time */
    iNdim = 2;
    iDim[0] = 1;
    iDim[1] = 1;
    pmxOut[5] = mxCreateNumericArray(iNdim,(const int*)iDim,mxSINGLE_CLASS,mxREAL);
    tt = (float*)mxGetData(pmxOut[5]);
    
    /* allocate host memory */
    /* bx1, bx2, bx3 */
    h_bx1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_bx2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_bx3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_bx1 || !h_bx2 || !h_bx3) mexPrintf("calloc: Memory allocation failure\n");
    
    /* by1, by2, by3 */
    h_by1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_by2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_by3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_by1 || !h_by2 || !h_by3) mexPrintf("calloc: Memory allocation failure\n");
    
    /* bz1, bz2, bz3 */
    h_bz1 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_bz2 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_bz3 = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_bz1 || !h_bz2 || !h_bz3) mexPrintf("calloc: Memory allocation failure\n");
    
    /* q */
    h_q = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_q) mexPrintf("calloc: Memory allocation failure\n");
    
    /* gk */
    h_gkx = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_gky = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_gkz = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_gkx || !h_gky || !h_gkz) mexPrintf("calloc: Memory allocation failure\n");
    
    /* div */
    h_dvx = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_dvy = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    h_dvz = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_dvx || !h_dvy || !h_dvz ) mexPrintf("calloc: Memory allocation failure\n");
    
    /* h_FPS */
    h_FPS = (float *) calloc( (unsigned)imageSize, sizeof(float) );
    if (!h_FPS) mexPrintf("calloc: Memory allocation failure\n");
    
    
    
    /* device memory allocation */
    hipMalloc( (void**) &d_bx1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_bx2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_bx3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_by1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_by2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_by3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_bz1, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_bz2, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_bz3, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_gkx, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_gky, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_gkz, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dvx, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dvy, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_dvz, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_q,  sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_ux, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_uy, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_uz, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_Ux, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_Uy, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_Uz, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_Gx, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_Gy, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_Gz, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_Gt, sizeof(float)*(unsigned)imageSize);
    hipMalloc( (void**) &d_Gf, sizeof(float)*(unsigned)imageSize);
    
    hipMalloc( (void**) &d_FPS, sizeof(float)*(unsigned)imageSize);
    
    /* copy arrays from host to device */
    hipMemcpy( d_bx1, h_bx1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_bx2, h_bx2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_bx3, h_bx3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_by1, h_by1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_by2, h_by2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_by3, h_by3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_bz1, h_bz1, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_bz2, h_bz2, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_bz3, h_bz3, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_gkx, h_gkx, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_gky, h_gky, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_gkz, h_gkz, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dvx, h_dvx, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dvy, h_dvy, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_dvz, h_dvz, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_q,  h_q,  sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_ux, h_ux, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_uy, h_uy, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_uz ,h_uz, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_Ux, h_Ux, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_Uy, h_Uy, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_Uz, h_Uz, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_Gx, h_Gx, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_Gy, h_Gy, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_Gz, h_Gz, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_Gt, h_Gt, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    hipMemcpy( d_Gf, h_Gf, sizeof(float)*(unsigned)imageSize, hipMemcpyHostToDevice);
    
    /* run optimization */
    
    /* iNI = 0; */
    dim3 threads(BLOCKSIZE,1,1);
    dim3 grid = GetGrid(imageSize);
    
    for( int i = 0; i < maxIter; i++){
        
        /* update p */
        krnl_1<<<grid, threads>>>(d_dvx, d_dvy, d_dvz, d_ux, d_uy, d_uz,
                d_gkx, d_gky, d_gkz, d_Gx, d_Gy, d_Gz,
                d_Gt, d_Gf, d_q, d_Ux, d_Uy, d_Uz,
                cc, iNx, iNy, iNz);

        /* update px, py, pz */
        krnl_23z<<<grid, threads>>>(d_bx1, d_by1, d_bz1, 
                d_bx2, d_by2, d_bz2,
                d_bx3, d_by3, d_bz3,
                d_gkx, d_gky, d_gkz,
                steps, iNx, iNy, iNz);
        /*
//         krnl_2<<<grid, threads>>>(d_bx1, d_by1, d_bz1, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
// 
//         krnl_3<<<grid, threads>>>(d_bx2, d_by2, d_bz2, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
//         
//         krnl_z<<<grid, threads>>>(d_bx3, d_by3, d_bz3, d_gkx, d_gky, d_gkz,
//                 steps, iNx, iNy, iNz);
        */
        
        /* projection step */
        krnl_4<<<grid, threads>>>(d_bx1, d_bx2, d_bx3, d_by1, d_by2, d_by3,
                d_bz1, d_bz2, d_bz3, d_gkx, d_gky, d_gkz,
                fPenalty, iNx, iNy, iNz);
        
        krnl_56zp<<<grid, threads>>>(d_bx1, d_by1, d_bz1,
                d_bx2, d_by2, d_bz2,
                d_bx3, d_by3, d_bz3,
                d_gkx, d_gky, d_gkz,
                iNx, iNy, iNz);
        /*
//         krnl_5<<<grid, threads>>>(d_bx1, d_by1, d_bz1, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
//         
//         krnl_6<<<grid, threads>>>(d_bx2, d_by2, d_bz2, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
//         
//         krnl_zp<<<grid, threads>>>(d_bx3, d_by3, d_bz3, d_gkx, d_gky, d_gkz,
//                 iNx, iNy, iNz);
        */
        krnl_7<<<grid, threads>>>(d_bx1, d_bx2, d_bx3, d_by1, d_by2, d_by3,
                d_bz1, d_bz2, d_bz3, d_dvx, d_dvy, d_dvz, d_Gx, d_Gy, d_Gz,
                d_q, d_ux, d_uy, d_uz, d_FPS,
                cc, iNx, iNy, iNz);
        
        /* compute convergence */
        hipMemcpy( h_FPS, d_FPS, sizeof(float)*unsigned(imageSize), hipMemcpyDeviceToHost);
        
        fps = 0;
        for (int j=0; j< imageSize; j++){
            fps += abs(h_FPS[j]);
        }
        
        h_cvg[i] = fps / (float)imageSize;
        
        if (h_cvg[i] <= fError){
            break; 
        }
        
        /*mexPrintf("cvg: %f\n",h_cvg[i]); */
        
        punum[0] = i+1;
        
    }
    
    /* copy arrays from device to host */
    hipMemcpy( h_ux, d_ux, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    hipMemcpy( h_uy, d_uy, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    hipMemcpy( h_uz, d_uz, sizeof(float)*(unsigned)(imageSize), hipMemcpyDeviceToHost);
    
    mexPrintf("number of iterations = %i\n",punum[0]);
    
    
    /* Free memory */
    free( (float *) h_bx1 );
    free( (float *) h_bx2 );
    free( (float *) h_bx3 );
    free( (float *) h_by1 );
    free( (float *) h_by2 );
    free( (float *) h_by3 );
    free( (float *) h_bz1 );
    free( (float *) h_bz2 );
    free( (float *) h_bz3 );
    free( (float *) h_gkx );
    free( (float *) h_gky );
    free( (float *) h_gkz );
    free( (float *) h_dvx );
    free( (float *) h_dvy );
    free( (float *) h_dvz );
    free( (float *) h_q );
    
    free( (float *) h_FPS );
    
    /*    Free GPU Memory */
    hipFree(d_bx1);
    hipFree(d_bx2);
    hipFree(d_bx3);
    hipFree(d_by1);
    hipFree(d_by2);
    hipFree(d_by3);
    hipFree(d_bz1);
    hipFree(d_bz2);
    hipFree(d_bz3);
    hipFree(d_gkx);
    hipFree(d_gky);
    hipFree(d_gkz);
    hipFree(d_dvx);
    hipFree(d_dvy);
    hipFree(d_dvz);
    
    hipFree(d_ux);
    hipFree(d_uy);
    hipFree(d_uz);
    hipFree(d_Ux);
    hipFree(d_Uy);
    hipFree(d_Uz);
    hipFree(d_Gx);
    hipFree(d_Gy);
    hipFree(d_Gz);
    hipFree(d_Gt);
    hipFree(d_Gf);
    hipFree(d_q);
    hipFree(d_FPS);
    
    /* CUDA event-based timer */
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    hipEventDestroy( start );
    hipEventDestroy( stop );
    
    
    tt[0] = time;
    
    mexPrintf("\nComputational Time for Dual Optimization = %.4f sec\n \n",tt[0]/1000000);
    
    
}
